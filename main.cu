#include "mul.h"
#include <iostream>
#include <vector>

int main(){
	unsigned int N{1024*1024*1024};
    std::size_t size = N * sizeof(float);

    // local data
	std::vector<float> A(N, 1.0);
	std::vector<float> B(N, 2.0);
	std::vector<float> C(N, -99.0);

    // allocate device data
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // copy local values to device (count is in bytes!)
    hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), size, hipMemcpyHostToDevice);

    // invoke kernel
	VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // copy device values to local (count is in bytes!)
    hipMemcpy(C.data(), d_C, size, hipMemcpyDeviceToHost);

    // free device data
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    exit(0);
    // print results
	for (const auto& c : C){
    	std::cout << c << std::endl;
	}
}